#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
    __constant__ float c[19] = { 0.0,0.0,0.0,0.0,2.0, 0.0,0.0,0.0,0.0,2.0, 0.0,0.0,0.0,0.0,2.0, -2.0,2.0,-2.0,4.0 }，add[4] = { -1.5f,-0.5f,0.5f,1.5f }, last[4] = { 2.5f,3.5f,4.5f,5.5f };
    __constant__ int ans[16] = { 0,1,1,0,1,0,0,1,1,0,0,1,0,1,1,0 };
    __device__ float relu(float x) { return x * (x > 0.0f); }
    __global__ void xor_kernel() {
        unsigned long long id = (unsigned long long)blockDim.x * blockIdx.x + threadIdx.x;
        float w[19]; for (int i = 0; i < 18; i++) w[i] = c[i] + add[(id >> (2 * i)) % 4];
        int err[4] = { 0,0,0,0 };
        for (int i = 0; i < 4; i++) {
            int j = 0;
            for (int x = 0; x < 16; x++) {
                bool x3 = x & 1, x2 = (x >> 1) & 1, x1 = (x >> 2) & 1, x0 = (x >> 3) & 1; bool y = ans[j]; j++;
                float a0 = relu(x0 * w[0] + x1 * w[1] + x2 * w[2] + x3 * w[3] + w[4]),
                    a1 = relu(x0 * w[5] + x1 * w[6] + x2 * w[7] + x3 * w[8] + w[9]),
                    a2 = relu(x0 * w[10] + x1 * w[11] + x2 * w[12] + x3 * w[13] + w[14]);
                float z = a0 * w[15] + a1 * w[16] + a2 * w[17] + last[i];
                err[i] += ((z >= 0) != y);
            }
        }
        if (err[0] == 0) printf("%.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f\n",
            w[0], w[1], w[2], w[3], w[4], w[5], w[6], w[7], w[8], w[9], w[10], w[11], w[12], w[13], w[14], w[15], w[16], w[17], last[0]);
        if (err[1] == 0) printf("%.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f\n",
            w[0], w[1], w[2], w[3], w[4], w[5], w[6], w[7], w[8], w[9], w[10], w[11], w[12], w[13], w[14], w[15], w[16], w[17], last[1]);
        if (err[2] == 0) printf("%.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f\n",
            w[0], w[1], w[2], w[3], w[4], w[5], w[6], w[7], w[8], w[9], w[10], w[11], w[12], w[13], w[14], w[15], w[16], w[17], last[2]);
        if (err[3] == 0) printf("%.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f %.1f\n",
            w[0], w[1], w[2], w[3], w[4], w[5], w[6], w[7], w[8], w[9], w[10], w[11], w[12], w[13], w[14], w[15], w[16], w[17], last[3]);
    }
    int main(void) {
        xor_kernel << <512, 1024 >> > ();
        hipEvent_t start, stop; hipEventCreate(&start); hipEventCreate(&stop); hipEventRecord(start, 0);
        xor_kernel << < 1 << 29, 128 >> > ();
        hipEventRecord(stop, 0); hipEventSynchronize(stop);
        float elapsed_time = 0.0f; hipEventElapsedTime(&elapsed_time, start, stop);
        printf("Kernel Elapsed Time: %.3f s\n", elapsed_time / 1000.0);
        return 0;
    }
